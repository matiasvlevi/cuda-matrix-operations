#include "hip/hip_runtime.h"
#include "dann.hpp"

void Dann::log() {
    this->logArch();

    // Log layers
    for (int i = 0; i < layers.size(); i++) {
        matrix::log(layers[i]->values, layers[i]->size, 1);
    }

    // Log weights
    for (int i = 0; i < weights.size(); i++) {
        matrix::log(weights[i]->values, weights[i]->rows, weights[i]->cols);
    }
}

void Dann::logArch() {
    char delim;
    // Log Architecture
    std::cout << "Architecture: {";

    for (int i = 0; i < layers.size(); i++) {
        delim = (i == 0) ? ' ' : ',';
        std::cout << delim << layers[i]->size; 
    }

    std::cout << " }" << std::endl;
}