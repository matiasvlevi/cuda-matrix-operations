#include "dann.hpp"

void Dann::registerDeviceMem() {
	// Register layer values
	for (int i = 0; i < layers.size(); i++) {
		hipMalloc(&(layers[i]->cuda_values), layers[i]->bsize);
    
		std::cout << "layer "<< i << " size: " << layers[i]->bsize << std::endl;
	}


    // Register weight values
	for (int i = 0; i < weights.size(); i++) {
		hipMalloc(&(weights[i]->cuda_values), weights[i]->bsize);

        hipMemcpy(
            weights[i]->cuda_values,
            weights[i]->values,
            weights[i]->bsize,
            hipMemcpyHostToDevice
        );
		
		std::cout << "weight "<< i << " size: " << weights[i]->bsize << std::endl;
	}
}
