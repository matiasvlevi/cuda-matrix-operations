#include "../src/matrix/matrix.cuh"

int main() {

	int N = 2; // A matrix columns and B matrix rows
	int M = 3; // A matrix rows
	int P = 1; // B matrix columns

	float a[M*N];
	Matrix::initRandomi_static(a, M, N);

	float b[P*N];
	Matrix::initRandomi_static(b, N, P);

	float c[P*M];
	Matrix::initRandomi_static(c, M, P);

	Matrix::log_static(a, M, N, 'A');
	Matrix::log_static(b, N, P, 'B');

	float *cudaA = 0;
	float *cudaB = 0;
	float *cudaC = 0;

	hipMalloc(&cudaA, sizeof(a));
 	hipMalloc(&cudaB, sizeof(b));
 	hipMalloc(&cudaC, sizeof(c));
 
 	hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
 	hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);
 	hipMemcpy(cudaC, c, sizeof(c), hipMemcpyHostToDevice);
 
 	dim3 THREADS;
 	THREADS.x = 16;
 	THREADS.y = 16;
 
 	int blocks = (N + THREADS.x - 1) / THREADS.x;
 
 	dim3 BLOCKS;
 	BLOCKS.x = blocks;
 	BLOCKS.y = blocks;
	
	std::cout << "Start" << std::endl;
	
	Kernel::dot<<<BLOCKS, THREADS>>>(cudaA, cudaB, cudaC, N, M, P);
 
	hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);
 	
 	hipFree(cudaA);
 	hipFree(cudaB);
 	hipFree(cudaC);
 
 	Matrix::log_static(c, M, P, 'C');

	std::cout << "Finished" << std::endl;

	return 0;
}
