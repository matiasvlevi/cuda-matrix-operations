#include "../matrix/matrix.cuh"

int main() {

	int N = 8; // matrix cols
	int M = 6; // matrix rows

	float a[M*N];
	Matrix::initRandomi_static(a, N, M);

	float c[N*M];
	Matrix::initRandomi_static(c, M, N);

	float *cudaA = 0;
	float *cudaC = 0;

	std::cout << "Start" << std::endl;
	
	Matrix::log_static(a, N, M, 'A');
 	
	hipMalloc(&cudaA, sizeof(a));
 	hipMalloc(&cudaC, sizeof(c));
 
 	hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
 	hipMemcpy(cudaC, c, sizeof(c), hipMemcpyHostToDevice);
 
 	dim3 THREADS;
 	THREADS.x = 16;
 	THREADS.y = 16;
 
 	int blocks = (N + THREADS.x - 1) / THREADS.x;
 
 	dim3 BLOCKS;
 	BLOCKS.x = blocks;
 	BLOCKS.y = blocks;
	
	Kernel::transpose<<<BLOCKS, THREADS>>>(cudaA, cudaC, N, M);
 
	hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);
 	
 	hipFree(cudaA);
 	hipFree(cudaC);
 
 	Matrix::log_static(c, M, N, 'C');

	std::cout << "Finished" << std::endl;

	return 0;
}
