#include "hip/hip_runtime.h"
#include "../matrix/matrix.cuh"


int main() {

	int N = 8; // matrix cols
	int M = 6; // matrix rows

	float a[M*N];
	matrix::init(a, N, M);

	float c[N*M];
	matrix::init(c, M, N);

	float *cudaA = 0;
	float *cudaC = 0;

	std::cout << "Start" << std::endl;
	
	matrix::log(a, N, M, 'A');
 	
	hipMalloc(&cudaA, sizeof(a));
 	hipMalloc(&cudaC, sizeof(c));
 
 	hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
 	hipMemcpy(cudaC, c, sizeof(c), hipMemcpyHostToDevice);
 
 	dim3 THREADS;
 	THREADS.x = 16;
 	THREADS.y = 16;
 
 	int blocks = (N + THREADS.x - 1) / THREADS.x;
 
 	dim3 BLOCKS;
 	BLOCKS.x = blocks;
 	BLOCKS.y = blocks;
	
	matrix::transpose<<<BLOCKS, THREADS>>>(cudaA, cudaC, N, M);
 
	hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);
 	
 	hipFree(cudaA);
 	hipFree(cudaC);
 
 	matrix::log(c, M, N, 'C');

	std::cout << "Finished" << std::endl;

	return 0;
}
