#include "../src/matrix/matrix.cuh"

int main() {
	int N = 6; // A matrix columns
	int M = 6; // A matrix rows

	float a[M*N];
	Matrix::initRandomi_static(a, M, N);

	float c[M*N];
	Matrix::init_static(c, M, N);

	Matrix::log_static(a, M, N, 'A');
	Matrix::log_static(c, M, N, 'C');

	float *cudaA = 0;
	float *cudaC = 0;

	hipMalloc(&cudaA, sizeof(a));
 	hipMalloc(&cudaC, sizeof(c));
 
 	hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
 	hipMemcpy(cudaC, c, sizeof(c), hipMemcpyHostToDevice);
	
	mathFunc h_pointFunc;

	hipError_t err = hipMemcpyFromSymbol(&h_pointFunc, HIP_SYMBOL(Activation::sigmoid), sizeof(mathFunc));
	
 	dim3 THREADS;
 	THREADS.x = 16;
 	THREADS.y = 16;
 
 	int blocks = (N + THREADS.x - 1) / THREADS.x;
 
 	dim3 BLOCKS;
 	BLOCKS.x = blocks;
 	BLOCKS.y = blocks;
	
	std::cout << "Start" << std::endl;
	
	Kernel::map<<<BLOCKS, THREADS>>>(cudaA, cudaC, M, N, h_pointFunc);
	
	hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);
 	
 	hipFree(cudaA);
 	hipFree(cudaC);

 	Matrix::log_static(c, M, N, 'C');

	std::cout << "Finished" << std::endl;

	return 0;
}
