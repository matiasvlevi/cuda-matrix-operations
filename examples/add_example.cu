#include "hip/hip_runtime.h"
#include "../matrix/matrix.cuh"

int main() {

	int N = 8; // matrix rows
	int M = 8; // matrix cols

	float a[M*N];
	matrix::init(a, M, N);

	float b[M*N];
	matrix::init(b, M, N);

	float c[M*N];
	matrix::init(c, M, N);

	float *cudaA = 0;
	float *cudaB = 0;
	float *cudaC = 0;

	std::cout << "Start" << std::endl;
	
	matrix::log(a, M, N, 'A');
	matrix::log(b, M, N, 'B');
 	
	hipMalloc(&cudaA, sizeof(a));
 	hipMalloc(&cudaB, sizeof(b));
 	hipMalloc(&cudaC, sizeof(c));
 
 	hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
 	hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);
 	hipMemcpy(cudaC, c, sizeof(c), hipMemcpyHostToDevice);
 
 	dim3 THREADS;
 	THREADS.x = 16;
 	THREADS.y = 16;
 
 	int blocks = (N + THREADS.x - 1) / THREADS.x;
 
 	dim3 BLOCKS;
 	BLOCKS.x = blocks;
 	BLOCKS.y = blocks;
	
	matrix::add<<<BLOCKS, THREADS>>>(cudaA, cudaB, cudaC, N, M);
 
	hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);
 	
 	hipFree(cudaA);
 	hipFree(cudaB);
 	hipFree(cudaC);
 
 	matrix::log(c, M, N, 'C');

	std::cout << "Finished" << std::endl;

	return 0;
}
