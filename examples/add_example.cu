#include "../matrix/matrix.cuh"

int main() {

	int N = 8; // matrix rows
	int M = 8; // matrix cols

	float a[M*N];
	Matrix::initRandomi_static(a, M, N);

	float b[M*N];
	Matrix::initRandomi_static(b, M, N);

	float c[M*N];
	Matrix::initRandomi_static(c, M, N);

	float *cudaA = 0;
	float *cudaB = 0;
	float *cudaC = 0;

	std::cout << "Start" << std::endl;
	
	Matrix::log_static(a, M, N, 'A');
	Matrix::log_static(b, M, N, 'B');
 	
	hipMalloc(&cudaA, sizeof(a));
 	hipMalloc(&cudaB, sizeof(b));
 	hipMalloc(&cudaC, sizeof(c));
 
 	hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
 	hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);
 	hipMemcpy(cudaC, c, sizeof(c), hipMemcpyHostToDevice);
 
 	dim3 THREADS;
 	THREADS.x = 16;
 	THREADS.y = 16;
 
 	int blocks = (N + THREADS.x - 1) / THREADS.x;
 
 	dim3 BLOCKS;
 	BLOCKS.x = blocks;
 	BLOCKS.y = blocks;
	
	Kernel::add<<<BLOCKS, THREADS>>>(cudaA, cudaB, cudaC, N, M);
 
	hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);
 	
 	hipFree(cudaA);
 	hipFree(cudaB);
 	hipFree(cudaC);
 
 	Matrix::log_static(c, M, N, 'C');

	std::cout << "Finished" << std::endl;

	return 0;
}
