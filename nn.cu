
#include <hip/hip_runtime.h>
#include <iostream>

/**
* matrixMult
* 
* Matrix dot product
*/
__global__ void matrixMult(float *a, float *b, float *c, int N, int M, int P) {

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;	

	// Abort if out of range
	if (row >= M || col >= P) return;

	float sum = 0;
	for (int k = 0; k < P; k++) {
		sum += a[row * N + k] * b[k * P + col];
	}

	c[row * P + col] = sum;

	return;
}

/**
* matrixAdd
* 
* Add 2 matrices with the same dimension
*
*/
__global__ void matrixAdd(float *a, float *b, float *c, int R, int C) {

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;	

	// Abort if out of range
	if (row >= R || col >= C) return;

	c[row * C + col] = a[row * C + col] + b[row * C + col];

	return;
}

/**
* makeMatrix
* 
* initiate a matrix with R,C dimensions
*
*/
void makeMatrix(float* m, int R, int C) {
	for (int i = 0; i < R*C; i++) {
		m[i] = (float)(rand() % 1000)/1000;		
	}
}

/**
* logMatrix
* 
* log a matrix in the console
* 
*/
void logMatrix(float* m, int R, int C) {
	char delim = ',';
	std::cout << "[\n";
	for (int i = 0; i < R; i++) {
		std::cout << "\t";
		for (int j = 0; j < C; j++) {
			delim = (j < C-1) ? ',' : ' ';
			std::cout << m[i * R + j] << delim;
		}
		std::cout << "\n";
	}


	std::cout << "]" << std::endl;
}

int main() {

	int N = 3;
	int M = 2;
	int P = 4;

	float a[M*N];
	makeMatrix(a, M, N);

	float b[P*N];
	makeMatrix(b, N, P);

	float c[P*M];
	makeMatrix(c, M, P);

	float *cudaA = 0;
	float *cudaB = 0;
	float *cudaC = 0;

	std::cout << "Start" << std::endl;
	logMatrix(a, M, N);
	logMatrix(b, N, P);
 	
	hipMalloc(&cudaA, sizeof(a));
 	hipMalloc(&cudaB, sizeof(b));
 	hipMalloc(&cudaC, sizeof(c));
 
 	hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
 	hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);
 	hipMemcpy(cudaC, c, sizeof(c), hipMemcpyHostToDevice);
 
 	dim3 THREADS;
 	THREADS.x = 16;
 	THREADS.y = 16;
 
 	int blocks = (N + THREADS.x - 1) / THREADS.x;
 
 	dim3 BLOCKS;
 	BLOCKS.x = blocks;
 	BLOCKS.y = blocks;
	
	matrixMult<<<BLOCKS, THREADS>>>(cudaA, cudaB, cudaC, N, M, P);
 
	hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);
 	
 	hipFree(cudaA);
 	hipFree(cudaB);
 	hipFree(cudaC);
 
 	logMatrix(c, M, P);

	std::cout << "Finished" << std::endl;

	return 0;
}
