#include "hip/hip_runtime.h"
#include "matrix/matrix.cuh"

int main() {

	int N = 3; // A matrix columns and B matrix rows
	int M = 2; // A matrix rows
	int P = 4; // B matrix columns

	float a[M*N];
	matrix::init(a, M, N);

	float b[P*N];
	matrix::init(b, N, P);

	float c[P*M];
	matrix::init(c, M, P);

	float *cudaA = 0;
	float *cudaB = 0;
	float *cudaC = 0;

	std::cout << "Start" << std::endl;
	
	matrix::log(a, M, N, 'A');
	matrix::log(b, N, P, 'B');
 	
	hipMalloc(&cudaA, sizeof(a));
 	hipMalloc(&cudaB, sizeof(b));
 	hipMalloc(&cudaC, sizeof(c));
 
 	hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
 	hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);
 	hipMemcpy(cudaC, c, sizeof(c), hipMemcpyHostToDevice);
 
 	dim3 THREADS;
 	THREADS.x = 16;
 	THREADS.y = 16;
 
 	int blocks = (N + THREADS.x - 1) / THREADS.x;
 
 	dim3 BLOCKS;
 	BLOCKS.x = blocks;
 	BLOCKS.y = blocks;
	
	matrix::dot<<<BLOCKS, THREADS>>>(cudaA, cudaB, cudaC, N, M, P);
 
	hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);
 	
 	hipFree(cudaA);
 	hipFree(cudaB);
 	hipFree(cudaC);
 
 	matrix::log(c, M, P, 'C');

	std::cout << "Finished" << std::endl;

	return 0;
}
