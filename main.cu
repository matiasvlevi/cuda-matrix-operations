#include "hip/hip_runtime.h"
#include "dann/dann.hpp"

int main() {
	Dann *nn = new Dann(2, 4);

	nn->addHiddenLayer(8);
	nn->makeWeights();
	nn->log();
	
	nn->registerDeviceMem();

    float input[2] = {1, 1};
	std::vector<float> out = nn->feedForward(input);

	Matrix::logVector(out);

	return 0;
}
