#include "hip/hip_runtime.h"
#include <iostream>

__global__ 
void addLists(int *a, int *b, int *c) {
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
	return;
}

int main() {

	int N = 12;

	int a[N] = { 2, 3, 1, 1, 4, 1, 3, 5, 2, 4, 4, 5 };
	int b[N] = { 1, 2, 5, 6, 2, 1, 4, 5, 6, 7, 8, 2 };
	int c[N] = { 0 };

	int *cudaA = 0;
	int *cudaB = 0;
	int *cudaC = 0;

	hipMalloc(&cudaA, sizeof(a));
	hipMalloc(&cudaB, sizeof(b));
	hipMalloc(&cudaC, sizeof(c));

	hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
	hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);
	hipMemcpy(cudaC, c, sizeof(c), hipMemcpyHostToDevice);

	addLists<<<1, N>>>(cudaA, cudaB, cudaC);
	
	hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);
	
	for (int i = 0; i < N; i++) 
		std::cout << c[i] << std::endl;

	return 0;
}
