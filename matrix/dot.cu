#include "hip/hip_runtime.h"
#include "matrix.cuh"

/**
* Mult
* 
*  dot product
*/
__global__ void matrix::dot(
    float *a,
    float *b,
    float *c,
    int N,
    int M,
    int P
) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;	

    // Abort if out of range
    if (row >= M || col >= P) return;

    float sum = 0;
    for (int k = 0; k < P; k++) {
        sum += a[row * N + k] * b[k * P + col];
    }

    c[row * P + col] = sum;

    return;
}