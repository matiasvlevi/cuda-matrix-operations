#include "hip/hip_runtime.h"
#include "matrix.cuh"

/**
* log
* 
* log a  in the console
* 
*/
void matrix::log(float* m, int R, int C, char name) {
    char delim = ',';
    std::cout << name << " [\n";
    for (int i = 0; i < R; i++) {
        std::cout << "\t";
        for (int j = 0; j < C; j++) {
            delim = (j < C-1) ? ',' : ' ';
            std::cout << m[i * C + j] << delim;
        }
        std::cout << "\n";
    }

    std::cout << "]\n" << std::endl;
}