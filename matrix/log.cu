#include "hip/hip_runtime.h"
#include "matrix.cuh"



/**
* log
* 
* log a  in the console
* 
*/
void Matrix::log_static(float* m, int R, int C, char name) {
    char delim = ',';
    std::cout << name << " [\n";
    for (int i = 0; i < R; i++) {
        std::cout << "\t";
        for (int j = 0; j < C; j++) {
            delim = (j < C-1) ? ',' : ' ';
            std::cout << m[i * C + j] << delim;
        }
        std::cout << "\n";
    }

    std::cout << "]\n" << std::endl;
}

/**
* log
* 
* log a  in the console
* 
*/
void Matrix::log(char name) {
    char delim = ',';
    std::cout << name << " [\n";
    for (int i = 0; i < rows; i++) {
        std::cout << "\t";
        for (int j = 0; j < cols; j++) {
            delim = (j == 0) ? ' ' : ',';
            std::cout << delim << values[i * cols + j];
        }
        std::cout << "\n";
    }

    std::cout << "]\n" << std::endl;
}

void Matrix::logVector(std::vector<float> input) {
    std::cout << "Vector [\n";
    
    for (int i = 0; i < input.size(); i++) 
        std::cout << "\t" << input[i] << ",\n";
    
    std::cout << "]" << std::endl;
}