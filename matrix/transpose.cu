#include "hip/hip_runtime.h"
#include "matrix.cuh"

/**
* transpose
* 
* transpose a matrix
*/
__global__ void matrix::transpose(
    float *a,
    float *c,
    int R,
    int C
) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Abort if out of range
    if (row >= R || col >= C) return; 

    c[col * R + row] = a[row * C + col];
}