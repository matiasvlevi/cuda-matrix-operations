#include "hip/hip_runtime.h"
#include "../matrix.cuh"

/**
* dot
* 
* A*B = C
* dot product
*/
__global__ void Kernel::dot(
    float *a,
    float *b,
    float *c,
    int N,
    int M,
    int P
) {

	int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;	

    // Abort if out of range
    if (row >= M || col >= P) return;

    // Vector dot product
    float sum = 0;
    for (int k = 0; k < N; k++) {
        sum += a[row * N + k] * b[k * P + col];
    }
    c[row * P + col] = sum;

    return;
}
