#include "hip/hip_runtime.h"
#include "../matrix.cuh"

/**
* add
* 
* Add 2 matrices with the same dimension
*
*/
__global__ void Kernel::add(
    float *a,
    float *b,
    float *c,
    int R,
    int C
) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;	

    // Abort if out of range
    if (row >= R || col >= C) return;

    c[row * C + col] = a[row * C + col] + b[row * C + col];

    return;
}