#include "hip/hip_runtime.h"
#include "../matrix.cuh"

__global__
void Kernel::map(
	float *a,
	float *c,
	int N,
	int M,
	mathFunc op
) {
	
	int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;	

	// Abort if out of range
	if (row >= N || col >= M) return;

	c[row * M + col] = (*op)(a[row * M + col]); 

}	
