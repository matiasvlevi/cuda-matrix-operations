#include "hip/hip_runtime.h"
#include "../matrix.cuh"

__global__
void Kernel::map(
	float *a,
	float *c,
	int N,
	int M,
	mathFunc operation
) {
	
	int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;	

	// Abort if out of range
	if (row >= N || col >= M) return;

	c[row * M + col] = Activation::sigmoid(a[row * M + col]); 

}	
