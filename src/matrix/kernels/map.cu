#include "hip/hip_runtime.h"
#include "../matrix.cuh"

__global__
void Kernel::map(
	float *a,
	float *c,
	int M,
	mathFunc operation
) {
	
	int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;	

	c[row * M + col] = operation(a[row * M + col]); 

}
