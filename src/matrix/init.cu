#include "hip/hip_runtime.h"
#include "matrix.cuh"

/**
* init
* 
* initiate a  with R,C dimensions
*
*/
void Matrix::init_static(float* m, int R, int C) {
    for (int i = 0; i < R*C; i++) {
        m[i] = 0;		
    }
}

/**
* init
* 
* initiate a  with R,C dimensions
*
*/
void Matrix::init() {
    for (int i = 0; i < rows*cols; i++) {
        values[i] = 0;		
    }
}


/**
* initRandomi
* 
* initiate a  with R,C dimensions
*
*/
void Matrix::initRandomi_static(float* m, int R, int C, int min, int max) {
    for (int i = 0; i < R*C; i++) {
        m[i] = (rand() % (max - min)) + min;		
    }
}

/**
* initRandomi
* 
* initiate a  with R,C dimensions
*
*/
void Matrix::initRandomi(int min, int max) {
    for (int i = 0; i < rows*cols; i++) {
        values[i] = (rand() % (max - min)) + min;		
    }
}

/**
* initRandomf
* 
* initiate a  with R,C dimensions
*
*/
void Matrix::initRandomf_static(float* m, int R, int C, float min, float max) {
    for (int i = 0; i < R*C; i++) {
        m[i] = ((float)rand()/(float)RAND_MAX) * (max-min) + min;	
    }
}

/**
* initRandomf
* 
* initiate a  with R,C dimensions
*
*/
void Matrix::initRandomf(float min, float max) {
    for (int i = 0; i < rows*cols; i++) {
        values[i] = ((float)rand()/(float)RAND_MAX) * (max-min) + min;	
    }
}
