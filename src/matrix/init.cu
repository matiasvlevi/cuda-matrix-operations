#include "hip/hip_runtime.h"
#include "matrix.cuh"

/**
* init
* 
* initiate a  with R,C dimensions
*
*/
void Matrix::init_static(float* m, int R, int C) {
    for (int i = 0; i < R*C; i++) {
        m[i] = 0;		
    }
}

/**
* init
* 
* initiate a  with R,C dimensions
*
*/
void Matrix::init() {
    for (int i = 0; i < rows*cols; i++) {
        values[i] = 0;		
    }
}


/**
* initRandomi
* 
* initiate a  with R,C dimensions
*
*/
void Matrix::initRandomi_static(float* m, int R, int C, int min, int max) {
    for (int i = 0; i < R*C; i++) {
        m[i] = (rand() % (max - min)) + min;		
    }
}

/**
* initRandomi
* 
* initiate a  with R,C dimensions
*
*/
void Matrix::initRandomi(int min, int max) {
    for (int i = 0; i < rows*cols; i++) {
        values[i] = (rand() % (max - min)) + min;		
    }
}

/**
* initRandomf
* 
* initiate a  with R,C dimensions
*
*/
void Matrix::initRandomf_static(float* m, int R, int C) {
    for (int i = 0; i < R*C; i++) {
        m[i] = (float)((rand() % (1 << 11)))/(1 << 10) - 1;		
    }
}

/**
* initRandomf
* 
* initiate a  with R,C dimensions
*
*/
void Matrix::initRandomf() {
    for (int i = 0; i < rows*cols; i++) {
        values[i] = (float)(rand() % (1 << 11))/(1 << 10) - 1;		
    }
}
