#include "dann.hpp"

void Dann::makeWeights() {

    // Create output layer
    float *output = (float*)malloc(sizeof(float) * output_size);
	if (heapOutOfMem(output)) return;

	Matrix::init_static(output, output_size, 1);
    layers.push_back(new Layer(output, output_size));

	// Create a bias layer
    float *bias_layer = (float*)malloc(sizeof(float) * output_size);
	if (heapOutOfMem(bias_layer)) return;

	Matrix::init_static(bias_layer, output_size, 1);
    biases.push_back(new Layer(bias_layer, output_size));

	// Initiate Device's pointer to the activation function
	mathFunc h_pointFunc;
	hipMemcpyFromSymbol(&h_pointFunc, HIP_SYMBOL(Activation::sigmoid), sizeof(mathFunc));	
	activations.push_back(h_pointFunc);

    // Weights
    for (int i = 1; i < layers.size(); i++) {
        float *hidden_w = (float*)malloc(sizeof(float) * layers[i]->size * layers[i-1]->size);
		if (heapOutOfMem(hidden_w)) return;
		
		Matrix::initRandomf_static(hidden_w, layers[i]->size, layers[i-1]->size);
        
		weights.push_back(
            new Matrix(hidden_w, layers[i]->size, layers[i-1]->size)
        );
    }

}
