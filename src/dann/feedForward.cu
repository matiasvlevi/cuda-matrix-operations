#include "dann.hpp"

std::vector<float> Dann::feedForward(float *input) {
	if (unsafe) {
		std::vector<float> output;
		return output;
	};

	layers[0]->values = input;

    hipMemcpy(
        layers[0]->cuda_values,
        layers[0]->values,
        layers[0]->bsize,
        hipMemcpyHostToDevice
    );

	int blocks;

    for (int i = 0; i < weights.size(); i++) {

        blocks = (weights[i]->cols + THREADS.x-1) / THREADS.x;

        BLOCKS.x = blocks;
        BLOCKS.y = blocks;

		Kernel::dot<<<BLOCKS, THREADS>>>(
            weights[i]->cuda_values,
            layers[i]->cuda_values,
            layers[i+1]->cuda_values,
            weights[i]->cols,
			weights[i]->rows,
			1
		);

		Kernel::add<<<BLOCKS, THREADS>>>(
			biases[i]->cuda_values,
			layers[i+1]->cuda_values,
			tempBuffer,
			biases[i]->size,
			1
		);

		hipMemcpy(layers[i+1]->cuda_values, tempBuffer, layers[i+1]->bsize, hipMemcpyDeviceToDevice);

		Kernel::map<<<BLOCKS, THREADS>>>(
			layers[i+1]->cuda_values,
			tempBuffer,
			layers[i+1]->size,
			1,
			activations[i]
		);

		hipMemcpy(layers[i+1]->cuda_values, tempBuffer, layers[i+1]->bsize, hipMemcpyDeviceToDevice);
	}

    hipMemcpy(
        layers[layers.size()-1]->values,
        layers[layers.size()-1]->cuda_values,
        layers[layers.size()-1]->bsize,
        hipMemcpyDeviceToHost
    );

    std::vector<float> output{
        layers[layers.size()-1]->values,
        layers[layers.size()-1]->values + layers[layers.size()-1]->size
    };

    return output;
}
