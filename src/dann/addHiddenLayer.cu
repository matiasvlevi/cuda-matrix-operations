#include "dann.hpp"

void Dann::addHiddenLayer(int layer_size, mathFunc *act) {
    // Create hidden layer
    float *layer = (float*)malloc(sizeof(float) * layer_size);
    Matrix::init_static(layer, layer_size, 1);

    layers.push_back(new Layer(layer, layer_size));
	

	mathFunc h_activation;
	hipMemcpyFromSymbol(&h_activation, HIP_SYMBOL(*act), sizeof(mathFunc));
	activations.push_back(h_activation);

	//activations.push_back(new ActFunc(act, act));

}
