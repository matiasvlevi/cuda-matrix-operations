#include "dann.hpp"

void Dann::addHiddenLayer(int layer_size, mathFunc *act) {
    // Create hidden layer
    float *layer = (float*)malloc(sizeof(float) * layer_size);
	if (heapOutOfMem(layer)) return;

	Matrix::init_static(layer, layer_size, 1);
    layers.push_back(new Layer(layer, layer_size));


	// Create a bias layer
	float *bias_layer = (float*)malloc(sizeof(float) * layer_size);
	if (heapOutOfMem(bias_layer)) return;

    Matrix::initRandomf_static(bias_layer, layer_size, 1);
	biases.push_back(new Layer(bias_layer, layer_size));


	// Initialize the Device's pointer to the activation function 
	mathFunc h_activation;
	hipMemcpyFromSymbol(&h_activation, HIP_SYMBOL(*act), sizeof(mathFunc));
	activations.push_back(h_activation);
}
