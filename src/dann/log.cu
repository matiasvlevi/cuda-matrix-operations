#include "hip/hip_runtime.h"
#include "dann.hpp"

void Dann::log() {
    this->logArch();

    // Log layers
    for (int i = 0; i < layers.size(); i++) {
        Matrix::log_static(layers[i]->values, layers[i]->size, 1);
    }

    // Log weights
    for (int i = 0; i < weights.size(); i++) {
        weights[i]->log();
    }
}

void Dann::logArch() {
    char delim;
    // Log Architecture
    std::cout << "Architecture: {";

    for (int i = 0; i < layers.size(); i++) {
        delim = (i == 0) ? ' ' : ',';
        std::cout << delim << layers[i]->size; 
    }

    std::cout << " }" << std::endl;
}