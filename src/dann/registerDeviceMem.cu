#include "dann.hpp"

void Dann::registerDeviceMem() {
	if (unsafe) return;

	// Register layer values
	for (int i = 0; i < layers.size(); i++) {
		hipMalloc(&(layers[i]->cuda_values), layers[i]->bsize);    
	}

    // Register weight values
	for (int i = 0; i < weights.size(); i++) {
		hipMalloc(&(weights[i]->cuda_values), weights[i]->bsize);

        hipMemcpy(
            weights[i]->cuda_values,
            weights[i]->values,
            weights[i]->bsize,
            hipMemcpyHostToDevice
        );	
	}
}
