#include "dann.hpp"

void Dann::registerDeviceMem() {
	if (unsafe) return;

	// Register layer values
	for (int i = 0; i < layers.size(); i++) {
		hipMalloc(&(layers[i]->cuda_values), layers[i]->bsize);    
	}

	// Register bias values
	for (int i = 0; i < biases.size(); i++) {
		hipMalloc(&(biases[i]->cuda_values), biases[i]->bsize);

		hipMemcpy(
			biases[i]->cuda_values,
			biases[i]->values,
			biases[i]->bsize,
			hipMemcpyHostToDevice
		);
	}

    // Register weight values
	for (int i = 0; i < weights.size(); i++) {
		hipMalloc(&(weights[i]->cuda_values), weights[i]->bsize);

        hipMemcpy(
            weights[i]->cuda_values,
            weights[i]->values,
            weights[i]->bsize,
            hipMemcpyHostToDevice
        );	
	}
}
