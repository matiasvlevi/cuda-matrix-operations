#include "dann.hpp"

void Dann::outputActivation(mathFunc *act) {
	mathFunc h_pointFunc;
	hipMemcpyFromSymbol(&h_pointFunc, HIP_SYMBOL(*act), sizeof(mathFunc));	
	activations[activations.size()-1] = h_pointFunc;
}
