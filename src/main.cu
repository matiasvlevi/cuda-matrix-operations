#include "hip/hip_runtime.h"
#include "dann/dann.hpp"

int main() {
	using namespace Activation;

	Dann *nn = new Dann(2, 4);

	nn->addHiddenLayer(8, &sigmoid);
	nn->addHiddenLayer(8, &sigmoid);

	nn->outputActivation(&sigmoid);
	nn->makeWeights();
	nn->log();

	nn->registerDeviceMem();

	float input[2] = {1, 1};
	std::vector<float> out = nn->feedForward(input);

	Matrix::logVector(out);
	
	return 0;
}
