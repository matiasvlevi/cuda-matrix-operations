#include "hip/hip_runtime.h"
#include "activations.hpp"

namespace Activation {
	__host__ __device__
	float linear(float x) {
		return x;
	}

	__host__ __device__
	float sigmoid(float x) {
		return 1 / (1 + exp(-x));
	}

	namespace Derivative {
		__host__ __device__
		float linear_d(float x) {
			return 1;
		}
	
		__host__ __device__
		float sigmoid_d(float x) {
			float x1 = 1 / (1 + exp(-x));
			return x1 * (1 - x1);
		}
	}
}

