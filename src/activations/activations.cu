#include "hip/hip_runtime.h"
#include "activations.hpp"

namespace Activation {
	__device__
	float linear(float x) {
		return x;
	}

	__device__
	float sigmoid(float x) {
		return 1 / (1 + exp(-x));
	}
	
	namespace Derivative {
		float linear_d(float x) {
			return 1;
		}
	
		float sigmoid_d(float x) {
			float x1 = 1 / (1 + exp(-x));
			return x1 * (1 - x1);
		}
	}
}

