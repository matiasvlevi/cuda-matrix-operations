#include "hip/hip_runtime.h"
#include "activations.hpp"

namespace ActPointer {
	__host__ __device__
	float linear(float x) {
		return x;
	}

	__host__ __device__
	float sigmoid(float x) {
		return 1 / (1 + exp(-x));
	}

	namespace Derivative {
		__host__ __device__
		float linear(float x) {
			return 1;
		}
	
		__host__ __device__
		float sigmoid(float x) {
			float x1 = 1 / (1 + exp(-x));
			return x1 * (1 - x1);
		}
	}
};

__device__ mathFunc Activation::sigmoid = ActPointer::sigmoid;
__device__ mathFunc Activation::linear = ActPointer::linear;
__device__ mathFunc Activation::Derivative::sigmoid = ActPointer::Derivative::sigmoid;
__device__ mathFunc Activation::Derivative::linear = ActPointer::Derivative::linear;
