#include "hip/hip_runtime.h"
#include "activations.hpp"

namespace Activation {
	float linear(float x) {
		return x;
	}

	float sigmoid(float x) {
		return 1 / (1 + exp(-x));
	}
	
	namespace Derivative {
		float linear_d(float x) {
			return 1;
		}
	
		float sigmoid_d(float x) {
			float x1 = 1 / (1 + exp(-x));
			return x1 * (1 - x1);
		}
	}
}

